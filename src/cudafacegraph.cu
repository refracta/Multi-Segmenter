#include "hip/hip_runtime.h"
#include "cudafacegraph.h"
#include "lockutils.hpp"

ParallelFaceGraph::ParallelFaceGraph(std::vector<Triangle>* triangles, DS_timer* timer) : FaceGraph(triangles, timer) {
    init();
}

ParallelFaceGraph::ParallelFaceGraph(std::vector<Triangle>* triangles) : FaceGraph(triangles) {
    init();
}

struct AdjacentNode {
    glm::vec3* vertex = nullptr;
    int* adjacents;
    int filled_index = 0;
} typedef AdjacentNode;

__global__ void cuda_union_find(std::vector<Triangle>* triangles, int triangle_idx, std::vector<int> adj_triangles, int* adjacents,int adjacents_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //각 삼각형의 root를 자신으로 초기화
    if(idx >= adjacents_size)
        return;

    // 맞닿아 있는 삼각형이,
    int adjacent_triangle = adjacents[idx];
    // 자기 자신이 아니고,
    // 원래의 삼각형과도 맞닿아 있으면 루트를 원래의 삼각형으로 지정.
    if (triangle_idx != adjacent_triangle && is_connected(triangles->at(triangle_idx), triangles->at(adjacent_triangle))) {
        if(adj_triangles[adjacent_triangle] > adj_triangles[triangle_idx])
            adj_triangles[adjacent_triangle] = adj_triangles[triangle_idx];
    }
}

void ParallelFaceGraph::init() {
    timer->onTimer(TIMER_FACEGRAPH_INIT_A);
    /* 변수 선언 */
    Vec3Hash hash_function;
    size_t vertex_size = triangles->size() * 3;
    omp_lock_t* locks = new_locks(vertex_size);
    // 해제 주의
    size_t** vertex_hash_list;

    /* 초기화 */
    vertex_hash_list = new size_t*[vertex_size];
    for (int i = 0; i < vertex_size; i++) {
        vertex_hash_list[i] = new size_t[3];
    }

    /* 해시 리스트 초기화 및 카운트 세기 */
    int* over_count_map = new int[vertex_size];
    std::fill_n(over_count_map, vertex_size, 0);
    // duplicated_vertex_key에 중복 vertex들의 triangle 포함 횟수 합산

    #pragma omp parallel for
    for (int i = 0; i < triangles->size(); i++) {
        for (int j = 0; j < 3; j++) {
            glm::vec3 vertex = triangles->at(i).vertex[j];
            size_t index = vertex_hash_list[i][j] = hash_function(vertex) % vertex_size;
            #pragma omp atomic
            over_count_map[index]++;
        }
    }

    int max_count = 0;
    // #pragma omp parallel for reduction(max:max_count)
    for (int i = 0; i < vertex_size; i++) {
        if (over_count_map[i] > max_count) {
            max_count = over_count_map[i];
        }
    }

    AdjacentNode* adjacent_nodes = new AdjacentNode[vertex_size];
    #pragma omp parallel for
    for (int i = 0; i < triangles->size(); i++) {
        for (int j = 0; j < 3; j++) {
            glm::vec3& vertex = triangles->at(i).vertex[j];
            size_t vertex_hash = vertex_hash_list[i][j];
            bool is_exist = false;
            AdjacentNode* node = &adjacent_nodes[vertex_hash];

            size_t locked_hash = vertex_hash;
            omp_set_lock(&locks[locked_hash]);
            while (node->vertex != nullptr) {
                glm::vec3* target = node->vertex;
                if (vertex == *target) {
                    is_exist = true;
                    break;
                }
                vertex_hash = (vertex_hash + 1) % vertex_size;
                node = &adjacent_nodes[vertex_hash];
            }
            vertex_hash_list[i][j] = vertex_hash;
            omp_unset_lock(&locks[locked_hash]);

            omp_set_lock(&locks[vertex_hash]);
            if (!is_exist) {
                node->vertex = &vertex;
                node->adjacents = new int[max_count];
                std::fill_n(node->adjacents, max_count, false);
            }
            node->adjacents[node->filled_index++] = i;
            omp_unset_lock(&locks[vertex_hash]);
        }
    }

    timer->offTimer(TIMER_FACEGRAPH_INIT_A);

    timer->onTimer(TIMER_FACEGRAPH_INIT_B);
    // 각 면에 대한 인접 리스트 생성.
    adj_triangles = std::vector<std::vector<int>>(triangles->size());
    triangles_parents = std::vector<int>(triangles->size());
    std::vector<int> dev_triangles_parents = std::vector<int>(triangles->size());
    std::vector<Triangle>* dev_triangles;

    //각 삼각형의 root를 자신으로 초기화
    #pragma omp parallel for
    for(int i = 0; i < triangles->size(); i++){
        triangles_parents[i] = i;
    }
    hipMemcpy(&dev_triangles_parents, &triangles_parents, triangles->size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&dev_triangles, &triangles, triangles->size() * sizeof(Triangle), hipMemcpyHostToDevice);

    // 각 삼각형에 대해서,
    for (int i = 0; i < triangles->size(); i++) {
        // 그 삼각형에 속한 정점과,
        for (int j = 0; j < 3; j++) {
            glm::vec3 vertex = triangles->at(i).vertex[j];

            size_t vertex_hash = vertex_hash_list[i][j];
            AdjacentNode* node = &adjacent_nodes[vertex_hash];

            int* adjacents = node->adjacents;
            int* dev_adjacents = 0;
            hipMemcpy(&dev_adjacents, &adjacents, node->filled_index * sizeof(int), hipMemcpyHostToDevice);

            dim3 dimBlock(1024, 1);
            dim3 dimGrid(ceil(node->filled_index / 1024), 1);
            cuda_union_find << < dimGrid, dimBlock >> > (dev_triangles, i, dev_triangles_parents, dev_adjacents, node->filled_index);

            hipMemcpy(&triangles_parents, &dev_triangles_parents, triangles->size() * sizeof(int), hipMemcpyDeviceToHost);
        }
    }

    for (int i = 0; i < vertex_size; i++) {
        delete[] vertex_hash_list[i];
    }
    delete[] vertex_hash_list;

    delete[] over_count_map;

    delete[] adjacent_nodes;

    destroy_locks(locks, vertex_size);
    timer->offTimer(TIMER_FACEGRAPH_INIT_B);
}
int value(int va) {
    return va;
}
std::vector<std::vector<Triangle>> ParallelFaceGraph::get_segments() {
    timer->onTimer(TIMER_FACEGRAPH_GET_SETMENTS_A);

    timer->offTimer(TIMER_FACEGRAPH_GET_SETMENTS_A);

    timer->onTimer(TIMER_FACEGRAPH_GET_SETMENTS_B);
    std::vector<std::vector<Triangle>> component_list(count);
    omp_lock_t* locks = new_locks(triangles_parents.size());
    #pragma omp parallel for
    for(int i = 0; triangles_parents.size(); i++){
        omp_set_lock(&locks[i]);
        component_list[triangles_parents[i]].push_back(triangles->data()[i]);
        omp_unset_lock(&locks[i]);
    }

    timer->offTimer(TIMER_FACEGRAPH_GET_SETMENTS_B);

    return component_list;
}

void ParallelFaceGraph::traverse_dfs(std::vector<int>& visit, int start_vert, int count) {
    std::stack<int> dfs_stack;
    dfs_stack.push(start_vert);

    while (!dfs_stack.empty()) {
        int current_vert = dfs_stack.top();
        dfs_stack.pop();

        visit[current_vert] = count;
        for (int i = 0; i < adj_triangles[current_vert].size(); i++) {
            int adjacent_triangle = adj_triangles[current_vert][i];
            if (visit[adjacent_triangle] == 0) {
                dfs_stack.push(adjacent_triangle);
            }
        }
    }
}
/*
void ParallelFaceGraph::union_find(std::vector<int>& visit, int start_vert, int count) {

    triangles_parents = std::vector<int>(triangles->size());

    //각 삼각형의 root를 자신으로 초기화
    #pragma omp parallel for
    for(int i = 0; i < triangles->size(); i++){
        triangles_parents[i] = i;
    }

    // 각 삼각형에 대해서,
    #pragma omp parallel for
    for (int i = 0; i < triangles->size(); i++) {
        // 그 삼각형에 속한 정점과,
        for (int j = 0; j < 3; j++) {
            glm::vec3 vertex = triangles->at(i).vertex[j];

            size_t vertex_hash = vertex_hash_list[i][j];
            AdjacentNode* node = &adjacent_nodes[vertex_hash];

            int* adjacents = node->adjacents;

            // 맞닿아 있는 삼각형이,
            for (int k = 0; k < node->filled_index; k++) {
                int adjacent_triangle = adjacents[k];
                // 자기 자신이 아니고,
                // 원래의 삼각형과도 맞닿아 있으면 루트를 원래의 삼각형으로 지정.
                if (i != adjacent_triangle && is_connected(triangles->at(i), triangles->at(adjacent_triangle))) {
                    if(triangles_parents[i] < triangles_parents[adjacent_triangle]){
                        triangles_parents[adjacent_triangle] = triangles_parents[i];    
                    }
                }
            }
        }
    }
}*/
