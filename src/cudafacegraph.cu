#include "hip/hip_runtime.h"
#include "cudafacegraph.h"

#define ADJ_MAX 20
#define BLOCK_LEN 512

__global__ void __get_vertex_to_adj(int* vertex_adj, size_t* triangles, int triangle_count, int* vertex_count_out, int vertex_index_begin, int adj_max) {
    __shared__ int vertex_index;
    __shared__ int index;
    __shared__ int internal_index;
    __shared__ int s_vertex_adj[ADJ_MAX];
    __shared__ glm::vec3 cache[BLOCK_LEN];

    if (threadIdx.x == 0) {
        vertex_index = vertex_index_begin + blockIdx.x;
        index = 0;
        internal_index = blockIdx.x;
    }
    __syncthreads();

    for (int j = 0; j < 3; j++) {
        for (int i = threadIdx.x; i < triangle_count; i += blockDim.x) {
            if (triangles[j * triangle_count + i] != vertex_index)
                continue;

            s_vertex_adj[atomicAdd(&index, 1)] = i;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int i = 0; i < index; i++)
            vertex_adj[internal_index * adj_max + i] = s_vertex_adj[i];
        vertex_count_out[internal_index] = index;
    }
}

std::vector<std::vector<int>> CUDAFaceGraph::get_vertex_to_adj() {
    std::vector<std::vector<int>> vertex_adjacent_map(total_vertex_count);

    // AOS에서 SOA로의 변환 비용이 없다고 가정.
    // 더 나은 변환 방법이 있거나 Triangle을 SOA로 변경하면 해결 가능.
    timer->offTimer(TIMER_FACEGRAPH_INIT_A);

    hipMalloc(&d_triangles_soa, triangles->size() * 3 * sizeof(size_t));
    for (int j = 0; j < 3; j++) {
        for (int i = 0; i < triangles->size(); i++) {
            hipMemcpy(&d_triangles_soa[j * triangles->size() + i], &triangles->at(i).id[j], sizeof(size_t), hipMemcpyHostToDevice);
        }
    }

    timer->onTimer(TIMER_FACEGRAPH_INIT_A);

    int adj_max = ADJ_MAX;
    int batch_size = 8192;
    int iter = (int)ceil((float)total_vertex_count / batch_size);

    std::vector<hipStream_t> streams(iter);

    for (hipStream_t& stream : streams)
        hipStreamCreate(&stream);

    std::vector<int*> vertex_adj(iter);
    std::vector<int*> vertex_count(iter);

    std::vector<int*> d_vertex_adj(iter);
    std::vector<int*> d_vertex_count(iter);

    // 동적 할당.
    for (int i = 0; i < iter; i++) {
        hipMallocAsync(&d_vertex_adj[i], batch_size * adj_max * sizeof(int), streams[i]);
        hipMallocAsync(&d_vertex_count[i], batch_size * sizeof(int), streams[i]);
    }

    // 동적 할당 host.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        hipHostMalloc(&vertex_adj[i], batch_size * adj_max * sizeof(int));
        hipHostMalloc(&vertex_count[i], batch_size * sizeof(int));
    }

    hipDeviceSynchronize();

    // 연산.
    for (int i = 0; i < iter; i++) {
        __get_vertex_to_adj<<<batch_size, std::min(triangles->size(), (size_t)BLOCK_LEN), 0, streams[i]>>>(d_vertex_adj[i],
                                                                                                           d_triangles_soa, triangles->size(),
                                                                                                           d_vertex_count[i], i * batch_size, adj_max);
    }
    hipDeviceSynchronize();

    // 데이터 복사 1.
    for (int i = 0; i < iter; i++) {
        hipMemcpyAsync(vertex_adj[i], d_vertex_adj[i], batch_size * adj_max * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        hipMemcpyAsync(vertex_count[i], d_vertex_count[i], batch_size * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
    }
    hipDeviceSynchronize();

    // 동적 할당 해제.
    for (int i = 0; i < iter; i++) {
        hipFreeAsync(&d_vertex_adj, streams[i]);
        hipFreeAsync(&d_vertex_count, streams[i]);
    }

    // 데이터 복사 2.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        for (int j = 0; j < batch_size; j++) {
            int index = i * batch_size + j;
            if (index < total_vertex_count)
                vertex_adjacent_map[index].insert(vertex_adjacent_map[index].begin(), vertex_adj[i] + (j * adj_max), vertex_adj[i] + (j * adj_max) + vertex_count[i][j]);
        }
    }

    // 동적 할당 해제 host.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        hipHostFree(&vertex_adj);
        hipHostFree(&vertex_count);
    }

    hipDeviceSynchronize();

    for (hipStream_t& stream : streams)
        hipStreamDestroy(stream);

    return vertex_adjacent_map;
}