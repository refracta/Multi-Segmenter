#include "hip/hip_runtime.h"
#include "cudafacegraph.h"

#define ADJ_MAX 20
#define BLOCK_LEN 512

__global__ void __get_vertex_to_adj(int* vertex_adj, Triangle* triangles, int triangle_count, int* vertex_count_out, int vertex_index_begin, int adj_max) {
    __shared__ int vertex_index;
    __shared__ int index;
    __shared__ int internal_index;
    __shared__ int s_vertex_adj[ADJ_MAX];
    __shared__ glm::vec3 cache[BLOCK_LEN];

    if (threadIdx.x == 0) {
        vertex_index = vertex_index_begin + blockIdx.x;
        index = 0;
        internal_index = blockIdx.x;
    }
    __syncthreads();

    for (int j = 0; j < 3; j++) {
        for (int i = threadIdx.x; i < triangle_count; i += blockDim.x) {
            if (triangles[i].id[j] != vertex_index)
                continue;

            s_vertex_adj[atomicAdd(&index, 1)] = i;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int i = 0; i < index; i++)
            vertex_adj[internal_index * adj_max + i] = s_vertex_adj[i];
        vertex_count_out[internal_index] = index;
    }
}

std::vector<std::vector<int>> CUDAFaceGraph::get_vertex_to_adj() {
    d_triangles = thrust::device_vector<Triangle>(*triangles);
    std::vector<std::vector<int>> vertex_adjacent_map(total_vertex_count);

    int adj_max = ADJ_MAX;
    int batch_size = 8192;
    int iter = (int)ceil((float)total_vertex_count / batch_size);

    std::vector<hipStream_t> streams(iter);

    for (hipStream_t& stream : streams)
        hipStreamCreate(&stream);

    std::vector<int*> vertex_adj(iter);
    std::vector<int*> vertex_count(iter);

    std::vector<int*> d_vertex_adj(iter);
    std::vector<int*> d_vertex_count(iter);

    // 동적 할당.
    for (int i = 0; i < iter; i++) {
        hipMallocAsync(&d_vertex_adj[i], batch_size * adj_max * sizeof(int), streams[i]);
        hipMallocAsync(&d_vertex_count[i], batch_size * sizeof(int), streams[i]);
    }

    // 동적 할당 host.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        hipHostMalloc(&vertex_adj[i], batch_size * adj_max * sizeof(int));
        hipHostMalloc(&vertex_count[i], batch_size * sizeof(int));
    }

    hipDeviceSynchronize();

    // 연산.
    for (int i = 0; i < iter; i++) {
        __get_vertex_to_adj<<<batch_size, std::min(triangles->size(), (size_t)BLOCK_LEN), 0, streams[i]>>>(d_vertex_adj[i],
                                                                                                           thrust::raw_pointer_cast(d_triangles.data()), d_triangles.size(),
                                                                                                           d_vertex_count[i], i * batch_size, adj_max);
    }
    hipDeviceSynchronize();

    // 데이터 복사 1.
    for (int i = 0; i < iter; i++) {
        hipMemcpyAsync(vertex_adj[i], d_vertex_adj[i], batch_size * adj_max * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        hipMemcpyAsync(vertex_count[i], d_vertex_count[i], batch_size * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
    }
    hipDeviceSynchronize();

    // 동적 할당 해제.
    for (int i = 0; i < iter; i++) {
        hipFreeAsync(&d_vertex_adj, streams[i]);
        hipFreeAsync(&d_vertex_count, streams[i]);
    }

    // 데이터 복사 2.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        for (int j = 0; j < batch_size; j++) {
            int index = i * batch_size + j;
            if (index < total_vertex_count)
                vertex_adjacent_map[index].insert(vertex_adjacent_map[index].begin(), vertex_adj[i] + (j * adj_max), vertex_adj[i] + (j * adj_max) + vertex_count[i][j]);
        }
    }

    // 동적 할당 해제 host.
    #pragma omp parallel for
    for (int i = 0; i < iter; i++) {
        hipHostFree(&vertex_adj);
        hipHostFree(&vertex_count);
    }

    hipDeviceSynchronize();

    for (hipStream_t& stream : streams)
        hipStreamDestroy(stream);

    return vertex_adjacent_map;
}