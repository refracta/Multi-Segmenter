#include "hip/hip_runtime.h"
﻿#include "cudafacegraph.h"
#include "cudasegmenter.h"

CUDASegmenter::CUDASegmenter(TriangleMesh* mesh, float tolerance) : Segmenter(mesh, tolerance) {
    timer.onTimer(TIMER_DATA_TRANSFER_D2H);
    deviceMesh = new DeviceTriangleMesh(mesh);
    timer.offTimer(TIMER_DATA_TRANSFER_D2H);
}

CUDASegmenter::~CUDASegmenter() {
    deviceMesh->free();
    free(deviceMesh);
}

inline glm::vec3 CUDASegmenter::get_normal_key(std::unordered_map<glm::vec3, size_t, Vec3Hash>& count_map,
                                               glm::vec3& normal) {
    for (const auto& entry : count_map) {
        glm::vec3 compare = entry.first;
        float norm_angle = glm::degrees(glm::angle(compare, normal));

        if (norm_angle < tolerance) {
            normal = compare;
            break;
        }
    }
    return normal;
}

inline void CUDASegmenter::init_count_map(std::unordered_map<glm::vec3, size_t, Vec3Hash>& count_map,
                                          std::vector<glm::vec3>& face_normals) {
    for (auto& normal : face_normals) {
        count_map[get_normal_key(count_map, normal)]++;
    }
}
struct NormalWrapper {
    glm::vec3 normal;
    float xAngle;
    float yAngle;
    float zAngle;
    Triangle triangle;
};

struct NormalMapper {
    glm::vec3* vertex;
    glm::vec3 xAxis = glm::vec3(1, 0, 0);
    glm::vec3 yAxis = glm::vec3(1, 0, 0);
    glm::vec3 zAxis = glm::vec3(1, 0, 0);

    explicit NormalMapper(glm::vec3* vertex) : vertex(vertex) {
    }

    __host__ __device__ NormalWrapper operator()(const glm::ivec3& idx) const {
        Triangle triangle;
        triangle.vertex[0] = vertex[idx[0]];
        triangle.vertex[1] = vertex[idx[1]];
        triangle.vertex[2] = vertex[idx[2]];
        glm::vec3 normal = glm::triangleNormal(triangle.vertex[0], triangle.vertex[1], triangle.vertex[2]);
        float xAngle = glm::angle(normal, xAxis);
        float yAngle = glm::angle(normal, yAxis);
        float zAngle = glm::angle(normal, zAxis);
        return {normal, xAngle, yAngle, zAngle, triangle};
    }
};

struct NormalIndexMapper {
    float tolerance;
    int baseSize;

    explicit NormalIndexMapper(float tolerance) {
        baseSize = floor(180.0f / tolerance);
        this->tolerance = glm::radians(tolerance);
    }

    __host__ __device__ int operator()(const NormalWrapper& normal) const {
        return ((int)floor(normal.xAngle / tolerance)) + ((int)floor(normal.yAngle / tolerance)) * baseSize +
               ((int)floor(normal.zAngle / tolerance)) * baseSize * baseSize;
    }
};

struct NormalTriangleMapper {
    explicit NormalTriangleMapper() {
    }

    __host__ __device__ Triangle operator()(const NormalWrapper& normal) const {
        return normal.triangle;
    }
};

struct AngleComparator {
    __host__ __device__ bool operator()(const NormalWrapper& o1, const NormalWrapper& o2) const {
        if (o1.xAngle < o2.xAngle)
            return true;
        else if (o1.xAngle > o2.xAngle)
            return false;

        // If xAngle is equal, then compare yAngle
        if (o1.yAngle < o2.yAngle)
            return true;
        else if (o1.yAngle > o2.yAngle)
            return false;

        // If yAngle is also equal, then compare zAngle
        if (o1.zAngle < o2.zAngle)
            return true;
        else
            return false;
    }
};

std::vector<TriangleMesh*> CUDASegmenter::do_segmentation() {
    timer.onTimer(TIMER_TOTAL);
    STEP_LOG(std::cout << "[Begin] Preprocessing.\n");
    timer.onTimer(TIMER_PREPROCESSING);
    STEP_LOG(std::cout << "[Begin] Normal Vector Computation.\n");
    timer.onTimer(TIMER_NORMAL_VECTOR_COMPUTATION);

    // obj에 포함된 면의 개수만큼 법선 벡터 계산 필요.
    thrust::device_vector<NormalWrapper> face_normals(mesh->index.size());
    thrust::transform(deviceMesh->index_device_vector->begin(), deviceMesh->index_device_vector->end(),
                      face_normals.begin(), NormalMapper(deviceMesh->vertex));

    timer.offTimer(TIMER_NORMAL_VECTOR_COMPUTATION);

    STEP_LOG(std::cout << "[End] Normal Vector Computation.\n");

    thrust::sort(face_normals.begin(), face_normals.end(), AngleComparator());
    thrust::device_vector<int> fn_indexes(face_normals.size());
    thrust::transform(face_normals.begin(), face_normals.end(), fn_indexes.begin(), NormalIndexMapper(tolerance));

    int baseSize = ceil(180.0f / tolerance);
    int binSize = baseSize * baseSize * baseSize;
    thrust::device_vector<int> indexes(binSize);
    thrust::device_vector<int> counts(binSize);
    thrust::reduce_by_key(fn_indexes.begin(), fn_indexes.end(), thrust::make_constant_iterator(1), indexes.begin(),
                          counts.begin(), thrust::equal_to<int>(), thrust::plus<int>());

    thrust::device_vector<Triangle> fn_triangles(face_normals.size());
    thrust::transform(face_normals.begin(), face_normals.end(), fn_triangles.begin(), NormalTriangleMapper());
    timer.offTimer(TIMER_PREPROCESSING);
    STEP_LOG(std::cout << "[End] Preprocessing.\n");

    STEP_LOG(std::cout << "[Begin] Connectivity Checking and Triangle Mesh Generating.\n");
    timer.onTimer(TIMER_CC_N_TMG);

    std::vector<int> startIndexes(binSize);
    int startIndex = 0;
    for (int i = 1; i < indexes.size(); i++) {
        startIndexes[i] = (startIndex += counts[i - 1]);
    }

    std::vector<TriangleMesh*> result;
    // 이제 병렬화가 가능할 것으로 보임
    int number = 0;
    for (int i = 0; i < binSize; i++) {
        int start = startIndexes[i];
        int end = start + counts[i];
        STEP_LOG(std::cout << "[Step] FaceGraph: Init.\n");
        std::vector<Triangle> triangles(counts[i]);
        thrust::copy(fn_triangles.begin() + start, fn_triangles.begin() + end, triangles.begin());
        // 오버헤드 제거 가능

        CUDAFaceGraph fg(&triangles, &timer);

        STEP_LOG(std::cout << "[Step] FaceGraph: Get Segments.\n");
        std::vector<std::vector<Triangle>> segments = fg.get_segments();

        STEP_LOG(std::cout << "[Step] Triangle Mesh Generating.\n");
        timer.onTimer(TIMER_TRIANGLE_MESH_GENERATING);
        for (const auto& segment : segments) {
            TriangleMesh* sub_object = triangle_list_to_obj(segment);
            sub_object->material->diffuse = glm::vec3(1, 0, 0);
            strcpy(sub_object->material->name, ("sub_materials_" + std::to_string(number)).c_str());
            strcpy(sub_object->name, (std::string(mesh->name) + "_seg_" + std::to_string(number++)).c_str());
            result.push_back(sub_object);
        }
        timer.offTimer(TIMER_TRIANGLE_MESH_GENERATING);
    }

    timer.offTimer(TIMER_CC_N_TMG);
    STEP_LOG(std::cout << "[End] Connectivity Checking and Triangle Mesh Generating.\n");

    STEP_LOG(std::cout << "[Begin] Segment Coloring.\n");
    timer.onTimer(TIMER_SEGMENT_COLORING);

    for (int i = 0; i < result.size(); i++) {
        result[i]->material->diffuse = Color::get_color_from_jet((float)i, 0, (float)result.size());
        result[i]->material->ambient = glm::vec3(1.0f, 1.0f, 1.0f);
        result[i]->material->specular = glm::vec3(0.5f, 0.5f, 0.5f);
    }

    STEP_LOG(std::cout << "[End] Segment Coloring.\n");
    timer.offTimer(TIMER_SEGMENT_COLORING);

    //    normal_triangle_list_map.clear();

    timer.offTimer(TIMER_TOTAL);
    return result;
};
