#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include ""
#include "hip/hip_runtime.h"
#include "grouping.h"
#include "trianglemesh.hpp"
#include <glm/gtx/normal.hpp>
#include <omp.h>
#include <cstdlib>
#include <algorithm>


#define BLOCK_SIZE 512
#define PI 3.14
#define SPLIT_SIZE 5

#define TIMER_PREPROCESSING 0
#define TIMER_NORMAL_VECTOR_COMPUTATION 1
#define TIMER_MAP_COUNT 2
#define TIMER_NORMAL_MAP_INSERTION 3
#define TIMER_TOTAL 11

struct Pair {
    unsigned int first;  // group id
    unsigned int second; // TriangleList index

    __host__ __device__ bool operator()(const Pair& a, const Pair& b) const {
        if (a.first < b.first)
            return true;
        return false;
    }
};

__global__ void grouping(Triangle* dVertexAlign, Pair* group, unsigned int indexSize, float tolerance,
                         unsigned int startPos) {

    unsigned int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
    unsigned int saveIndex = startPos + threadIdx.x + (blockIdx.x * blockDim.x);
    if (saveIndex >= indexSize)
        return;
    glm::vec3 normal = glm::normalize(glm::triangleNormal(
        dVertexAlign[threadId].vertex[0], dVertexAlign[threadId].vertex[1], dVertexAlign[threadId].vertex[2]));

    float xSeta = acosf(normal.z) / PI * 180;
    if (normal.z < 0.5f) // precision problem
        xSeta = 360 - xSeta;
    float ySeta = acosf(normal.x) / PI * 180;
    if (normal.x < 0.5f)
        ySeta = 360 - ySeta;
    float zSeta = acosf(normal.y) / PI * 180;
    if (normal.y < 0.5f)
        zSeta = 360 - zSeta;

    xSeta += 15; // 절대 각도 시작 위치 설정.
    ySeta += 15;
    zSeta += 15;

    unsigned int bitmap = (unsigned int)(xSeta / tolerance) % 360;
    bitmap = bitmap << 8;
    bitmap += (unsigned int)(ySeta / tolerance) % 360;
    bitmap = bitmap << 8;
    bitmap += (unsigned int)(zSeta / tolerance) % 360;

    group[saveIndex].first = bitmap;
    group[saveIndex].second = saveIndex;
}

__global__ void splitIndex(Pair* group, unsigned int* posList, unsigned int* size, unsigned int indexSize) {

    unsigned int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (threadId >= indexSize || threadId == 0)
        return;

    if (group[threadId].first != group[threadId - 1].first) {
        unsigned int prev = atomicAdd(size, 1);
        posList[prev] = threadId;
    }
}

std::unordered_map<unsigned int, std::vector<Triangle>> kernelCall(TriangleMesh* mesh, float tolerance,
                                                                   DS_timer& timer) {

    
    timer.onTimer(TIMER_PREPROCESSING);


    hipStream_t streamForAlloc;
    hipStreamCreate(&streamForAlloc);
    hipStream_t streamForCopy;
    hipStreamCreate(&streamForCopy);

    hipEvent_t eventListForAlloc[SPLIT_SIZE];
    hipEvent_t eventListForCopy[SPLIT_SIZE];
    for (int i = 0; i < SPLIT_SIZE; i++) {
        hipEventCreate(&eventListForAlloc[i]);
        hipEventCreate(&eventListForCopy[i]);
    }

    size_t indexSize = mesh->index.size();
    size_t calcSize = ceil((double)indexSize / SPLIT_SIZE);

    std::unordered_map<unsigned int, std::vector<Triangle>> normal_triangle_list_map;
    std::vector<Pair> hostData(indexSize);
    Triangle* dVertexAlign[SPLIT_SIZE];
    Pair* dGroup;
    Triangle* TriangleList = (Triangle*)malloc(sizeof(Triangle) * indexSize);
    Pair* group = (Pair*)malloc(sizeof(Pair) * indexSize);
    unsigned int* posList;
    unsigned int* dPos;
    unsigned int* dPosList;
    unsigned int pos = 0;

    // ------------------------------------- variable initial

    hipMallocAsync(&dGroup, sizeof(Pair) * indexSize, streamForAlloc);
    for (int i = 0; i < SPLIT_SIZE; i++) {
        hipMallocAsync(&dVertexAlign[i], sizeof(Triangle) * calcSize, streamForAlloc);
        hipEventRecord(eventListForAlloc[i], streamForAlloc);
    }
    hipMallocAsync(&dPos, sizeof(unsigned int), streamForAlloc);
    hipMallocAsync(&dPosList, sizeof(unsigned int) * pow(360.f / tolerance, 3), streamForAlloc);
    hipMemsetAsync(dPos, 0, sizeof(unsigned int), streamForAlloc);
    hipMemsetAsync(dPosList, 0, sizeof(unsigned int) * pow(360.f / tolerance, 3), streamForAlloc);

    // ------------------------------------ Triangle Caculate

    timer.onTimer(TIMER_NORMAL_VECTOR_COMPUTATION);

#pragma omp parallel for
    for (int i = 0; i < indexSize; i++) {
        TriangleList[i].vertex[0] = mesh->vertex[mesh->index[i].x];
        TriangleList[i].vertex[1] = mesh->vertex[mesh->index[i].y];
        TriangleList[i].vertex[2] = mesh->vertex[mesh->index[i].z];
    }

    // ----------------------------------- Vector Computation(grouping)

    size_t memCpyStart = 0;
    size_t groupingStart = 0;
    while (true) {

        for (size_t i = memCpyStart; i < SPLIT_SIZE; i++) {
            if (hipEventQuery(eventListForAlloc[i]) == hipSuccess) {
                if (i != SPLIT_SIZE - 1)
                    hipMemcpyAsync(dVertexAlign[i], &TriangleList[calcSize * i], sizeof(Triangle) * calcSize,
                                    hipMemcpyHostToDevice, streamForCopy);
                else
                    hipMemcpyAsync(dVertexAlign[i], &TriangleList[calcSize * i],
                                    sizeof(Triangle) * mesh->index.size() % SPLIT_SIZE, hipMemcpyHostToDevice,
                                    streamForCopy);

                hipEventRecord(eventListForCopy[i], streamForCopy);
                memCpyStart = i;
                if (memCpyStart == SPLIT_SIZE - 1)
                    memCpyStart++;
            } else
                break;
        }

        for (size_t i = groupingStart; i < SPLIT_SIZE; i++) {
            if (hipEventQuery(eventListForCopy[i]) == hipSuccess) {
                grouping<<<ceil((float)calcSize / BLOCK_SIZE), BLOCK_SIZE>>>(dVertexAlign[i], dGroup, indexSize,
                                                                             tolerance, calcSize * i);
                groupingStart = i;
                if (groupingStart == SPLIT_SIZE - 1)
                    groupingStart++;
            } else
                break;
        }

        if (memCpyStart == SPLIT_SIZE && groupingStart == SPLIT_SIZE)
            break;
    }

    hipStreamSynchronize(0);

    timer.offTimer(TIMER_NORMAL_VECTOR_COMPUTATION);

    // ----------------------------------- Sort

    timer.onTimer(TIMER_MAP_COUNT);

    hipStreamSynchronize(streamForAlloc);
    thrust::device_vector<Pair> deviceData(dGroup, dGroup + indexSize);
    thrust::sort(deviceData.begin(), deviceData.end(), Pair());
    thrust::copy(deviceData.begin(), deviceData.end(), hostData.begin());

    splitIndex<<<ceil((float)indexSize / BLOCK_SIZE), BLOCK_SIZE>>>(thrust::raw_pointer_cast(deviceData.data()),
                                                                    dPosList, dPos, indexSize);

    hipStreamSynchronize(0);

    posList = (unsigned int*)malloc(sizeof(unsigned int) * pow(360.f / tolerance, 3));

    hipMemcpy(posList, dPosList, sizeof(unsigned int) * pow(360.f / tolerance, 3), hipMemcpyDeviceToHost);
    hipMemcpy(&pos, dPos, sizeof(unsigned int), hipMemcpyDeviceToHost);

    posList[pos] = 0;
    pos++;
    posList[pos] = indexSize;
    pos++;

    std::sort(posList, posList + pos);

    timer.offTimer(TIMER_MAP_COUNT);

    // --------------------------------- Map Insertion

    timer.onTimer(TIMER_NORMAL_MAP_INSERTION);

    for (int i = 0; i < pos - 1; i++) {
        unsigned int start = posList[i];
        unsigned int end = posList[i + 1];
        unsigned int gid = hostData[start].first;
        normal_triangle_list_map.insert({gid, std::vector<Triangle>(end - start)});
    }

#pragma omp parallel for
    for (int i = 0; i < pos - 1; i++) {
        unsigned int start = posList[i];
        unsigned int end = posList[i + 1];
        unsigned int gid = hostData[start].first;

        for (unsigned int j = start; j < end; j++)
            normal_triangle_list_map[gid][j - start] = TriangleList[hostData[j].second];
    }

    timer.offTimer(TIMER_NORMAL_MAP_INSERTION);

    
    for (int i = 0; i < SPLIT_SIZE; i++) {
        hipFree(dVertexAlign[i]);
        hipEventDestroy(eventListForAlloc[i]);
        hipEventDestroy(eventListForCopy[i]);
    }
    hipStreamDestroy(streamForAlloc);
    hipStreamDestroy(streamForCopy);
    hipFree(dGroup);
    hipFree(dPos);
    hipFree(dPosList);
    free(TriangleList);
    free(group);
    free(posList);

    timer.offTimer(TIMER_PREPROCESSING);

    return normal_triangle_list_map;
}
