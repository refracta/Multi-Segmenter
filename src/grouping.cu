#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "grouping.h"
#include "trianglemesh.hpp"
#include <dstimer.hpp>
#include <glm/gtx/normal.hpp>
#include <omp.h>
#include <stdlib.h>

#include <algorithm>

struct Pair {
    unsigned int first;  // group id
    unsigned int second; // TriangleList index

    __device__ Pair& operator=(const Pair& other) {
        if (this != &other) { // protect against invalid self-assignment
            first = other.first;
            second = other.second;
        }
        // by convention, always return *this
        return *this;
    }

    __device__ bool operator()(const Pair& a, const Pair& b) const {
        if (a.first < b.first)
            return true;
        return false;
    }
};

#define PI 3.14
__global__ void grouping(Triangle* dVertexAlign, Pair* group, unsigned int indexSize, float tolerance) {

}

__global__ void splitIndex(Pair* group, unsigned int* posList, unsigned int* size, unsigned int indexSize) {

    unsigned int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (threadId >= indexSize || threadId == 0)
        return;

    if (group[threadId].first != group[threadId - 1].first) {
        unsigned int prev = atomicAdd(size, 1);
        posList[prev] = threadId;
    }
}

std::unordered_map<unsigned int, std::vector<Triangle>> kernelCall(TriangleMesh* mesh, float tolerance) {
    hipStream_t stream;
    hipStreamCreate(&stream);

    std::unordered_map<unsigned int, std::vector<Triangle>> normal_triangle_list_map;

    return normal_triangle_list_map;
}
