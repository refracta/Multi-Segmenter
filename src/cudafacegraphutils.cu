#include "hip/hip_runtime.h"
#include "cudafacegraphutils.h"

// cuda 관련 헤더를 .h 등 .cu가 아닌 파일에서 include하면 에러 발생.
#include <cuda/semaphore>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__global__ void __segment_union_to_obj(glm::vec3* vertices, glm::ivec3* faces, int* group_id, Triangle* triangles,
                                       size_t triangles_count, size_t total_vertex_count, int* index_lookup_chunk, int g_id,
                                       int* vertex_index_out, int* index_index_out) {
    __shared__ int vertex_index;    // push_back 대신 유지하는 정점 인덱스 추적 변수.
    __shared__ int index_index;     // push_back 대신 유지하는 삼각형 인덱스 추적 변수.
    __shared__ int* index_lookup;   // 기존 unordered_map을 유지하는 중복 검사용 변수.
    __shared__ cuda::binary_semaphore<cuda::thread_scope_block>* vertex_sem;     // 정점 삽입 mutex.

    if (threadIdx.x == 0) {
        vertex_index = 0;
        index_index = 0;
        index_lookup = index_lookup_chunk;
        vertex_sem = new cuda::binary_semaphore<cuda::thread_scope_block>();
        vertex_sem->release();
    }
    __syncthreads();

    for (int i = threadIdx.x; i < triangles_count; i += blockDim.x) {
        if (group_id[i] != g_id)
            continue;

        glm::ivec3 new_index;
        for (int j = 0; j < 3; j++) {
            int& index_if_exist = index_lookup[triangles[i].id[j]];

            vertex_sem->acquire();
            if (index_if_exist == -1) {
                vertices[vertex_index] = triangles[i].vertex[j];
                index_if_exist = ++vertex_index;
            }
            vertex_sem->release();

            new_index[j] = index_if_exist;
        }

        faces[atomicAdd(&index_index, 1) - 1] = new_index;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        *vertex_index_out = vertex_index;
        *index_index_out = index_index;
        delete vertex_sem;
    }
}

std::vector<TriangleMesh*> segment_union_to_obj(const std::vector<int> segment_union,
                                                const std::vector<Triangle>* triangles, size_t total_vertex_count) {
    std::vector<TriangleMesh*> result;
    std::vector<int> group_id(segment_union.size(), -1);    // 특정 요소가 속한 그룹 id.
    std::vector<int> group_count;                           // 특정 그룹의 요소 개수.

    int group_index = 0;
    for (int i = 0; i < segment_union.size(); i++) {
        int group_root = segment_union[i];
        int& g_id = group_id[group_root];

        if (g_id == -1) {
            result.push_back(new TriangleMesh);
            g_id = group_index++;
            result[g_id]->material = new Material;
            group_count.push_back(1);
        }

        group_id[i] = g_id;
        group_count[g_id]++;
    }

    std::vector<hipStream_t> streams(group_index);

    // cuda host.
    std::vector<glm::vec3*> vertex_out(group_index);
    std::vector<glm::ivec3*> face_out(group_index);
    int* vertex_index_out;
    int* face_index_out;

    for (int i = 0; i < group_index; i++) hipHostMalloc(&vertex_out[i], triangles->size() * 3 * sizeof(glm::vec3));
    for (int i = 0; i < group_index; i++) hipHostMalloc(&face_out[i], triangles->size() * sizeof(glm::ivec3));
    hipHostMalloc(&vertex_index_out, group_index * sizeof(int));
    hipHostMalloc(&face_index_out, group_index * sizeof(int));

    // 공통.
    thrust::device_vector<int> d_group_id(group_id);
    thrust::device_vector<Triangle> d_triangles(*triangles);

    for (hipStream_t& stream : streams)
        hipStreamCreate(&stream);

    #pragma omp parallel for
    for (int i = 0; i < group_index; i++) {
        hipStream_t& stream = streams[i];

        glm::vec3* d_vertices;
        glm::ivec3* d_faces;

        int* d_index_lookup;
        int* d_vertex_index_out;
        int* d_face_index_out;

        hipMallocAsync(&d_vertices, triangles->size() * 3 * sizeof(glm::vec3), stream);
        hipMallocAsync(&d_faces, triangles->size() * sizeof(glm::ivec3), stream);

        hipMallocAsync(&d_index_lookup, total_vertex_count * sizeof(int), stream);
        hipMallocAsync(&d_vertex_index_out, sizeof(int), stream);
        hipMallocAsync(&d_face_index_out, sizeof(int), stream);
        hipStreamSynchronize(stream);      // 동적 할당 동기화.

        hipMemsetAsync(d_index_lookup, 0xFF, total_vertex_count * sizeof(int), stream);
        hipStreamSynchronize(stream);      // 메모리 초기화 동기화.

        __segment_union_to_obj<<<1, std::min(triangles->size(), (size_t)1024), 0, stream>>>(d_vertices,
                                                                                            d_faces,
                                                                                            thrust::raw_pointer_cast(d_group_id.data()),
                                                                                            thrust::raw_pointer_cast(d_triangles.data()),
                                                                                            d_triangles.size(), total_vertex_count,
                                                                                            d_index_lookup,
                                                                                            i,
                                                                                            d_vertex_index_out,
                                                                                            d_face_index_out);
        hipStreamSynchronize(stream);      // 연산 동기화.

        hipMemcpyAsync(&vertex_index_out[i], d_vertex_index_out, sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(&face_index_out[i], d_face_index_out, sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);      // device -> host 동기화.

        hipMemcpyAsync(vertex_out[i], d_vertices, vertex_index_out[i] * sizeof(glm::vec3), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(face_out[i], d_faces, face_index_out[i] * sizeof(glm::ivec3), hipMemcpyDeviceToHost, stream);

        hipFreeAsync(d_vertices, stream);
        hipFreeAsync(d_faces, stream);

        hipFreeAsync(d_index_lookup, stream);
        hipFreeAsync(d_vertex_index_out, stream);
        hipFreeAsync(d_face_index_out, stream);
    }
    hipDeviceSynchronize();      // 최종 동기화.

    for (hipStream_t& stream : streams)
        hipStreamDestroy(stream);

    for (int i = 0; i < result.size(); i++) {
        result[i]->vertex.insert(result[i]->vertex.begin(), vertex_out[i], vertex_out[i] + vertex_index_out[i]);
        result[i]->index.insert(result[i]->index.begin(), face_out[i], face_out[i] + face_index_out[i]);
    }

    for (int i = 0; i < group_index; i++) hipHostFree(&vertex_out[i]);
    for (int i = 0; i < group_index; i++) hipHostFree(&face_out[i]);
    hipHostFree(vertex_index_out);
    hipHostFree(face_index_out);

    return result;
}
