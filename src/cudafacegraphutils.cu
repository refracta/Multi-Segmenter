#include "hip/hip_runtime.h"
#include "cudafacegraphutils.h"

// cuda 관련 헤더를 .h 등 .cu가 아닌 파일에서 include하면 에러 발생.
#include <cuda/semaphore>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__global__ void __segment_union_to_obj(glm::vec3* vertices, glm::ivec3* faces, int* group_id, Triangle* triangles,
                                       size_t triangles_count, size_t total_vertex_count, int* index_lookup_chunk, int __temp,
                                       int* vertex_index_out, int* index_index_out) {
    __shared__ int vertex_index;    // push_back 대신 유지하는 정점 인덱스 추적 변수.
    __shared__ int index_index;     // push_back 대신 유지하는 삼각형 인덱스 추적 변수.
    __shared__ cuda::binary_semaphore<cuda::thread_scope_block>* vertex_sem;     // 정점 삽입 mutex.

    __shared__ glm::vec3* local_vertices;
    __shared__ glm::ivec3* local_faces;
    __shared__ int* local_index_lookup;
    __shared__ int* local_vertex_index_out;
    __shared__ int* local_face_index_out;

    int g_id = blockIdx.x;

    if (threadIdx.x == 0) {
        vertex_index = 0;
        index_index = 0;
        vertex_sem = new cuda::binary_semaphore<cuda::thread_scope_block>();
        vertex_sem->release();

        local_vertices = &vertices[g_id * (triangles_count + 3)];
        local_faces = &faces[g_id * triangles_count];
        local_index_lookup = &index_lookup_chunk[g_id * total_vertex_count];
        local_vertex_index_out = &vertex_index_out[g_id];
        local_face_index_out = &index_index_out[g_id];
    }
    __syncthreads();

    for (int i = threadIdx.x; i < triangles_count; i += blockDim.x) {
        if (group_id[i] != g_id)
            continue;

        glm::ivec3 new_index;
        for (int j = 0; j < 3; j++) {
            int& index_if_exist = local_index_lookup[triangles[i].id[j]];

            vertex_sem->acquire();
            if (index_if_exist == -1) {
                local_vertices[vertex_index] = triangles[i].vertex[j];
                index_if_exist = ++vertex_index;
            }
            vertex_sem->release();

            new_index[j] = index_if_exist;
        }

        local_faces[atomicAdd(&index_index, 1)] = new_index;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        *local_vertex_index_out = vertex_index;
        *local_face_index_out = index_index;
        delete vertex_sem;
    }
}

std::vector<TriangleMesh*> segment_union_to_obj(const SegmentUnion segment_union,
                                                const std::vector<Triangle>* triangles, size_t total_vertex_count) {
    const std::vector<int>& group_id = segment_union.segment_union;
    int group_index = segment_union.group_count;

    std::vector<TriangleMesh*> result;
    for (int i = 0; i < group_index; i++) {
        result.push_back(new TriangleMesh);
        result[i]->material = new Material;
    }

    hipStream_t memset_stream;
    hipStreamCreate(&memset_stream);

    int* d_index_lookup; hipMalloc(&d_index_lookup, group_index * total_vertex_count * sizeof(int));
    hipMemsetAsync(d_index_lookup, 0xFF, group_index * total_vertex_count * sizeof(int), memset_stream);

    int* d_vertex_index_out; hipMalloc(&d_vertex_index_out, group_index * sizeof(int));
    int* d_face_index_out; hipMalloc(&d_face_index_out, group_index * sizeof(int));

    glm::vec3* d_vertices; hipMalloc(&d_vertices, group_index * triangles->size() * 3 * sizeof(glm::vec3));
    glm::ivec3* d_faces; hipMalloc(&d_faces, group_index * triangles->size() * sizeof(glm::ivec3));

    thrust::device_vector<int> d_group_id_vec(group_id);
    thrust::device_vector<Triangle> d_triangles_vec(*triangles);

    int* d_group_id = thrust::raw_pointer_cast(d_group_id_vec.data());
    Triangle* d_triangles = thrust::raw_pointer_cast(d_triangles_vec.data());

    std::vector<glm::vec3*> vertex_out(group_index);
    std::vector<glm::ivec3*> face_out(group_index);
    int* vertex_index_out;
    int* face_index_out;

    for (int i = 0; i < group_index; i++) hipHostMalloc(&vertex_out[i], triangles->size() * 3 * sizeof(glm::vec3));
    for (int i = 0; i < group_index; i++) hipHostMalloc(&face_out[i], triangles->size() * sizeof(glm::ivec3));
    hipHostMalloc(&vertex_index_out, group_index * sizeof(int));
    hipHostMalloc(&face_index_out, group_index * sizeof(int));

    hipStreamSynchronize(memset_stream);
    __segment_union_to_obj<<<group_index, std::min(triangles->size(), (size_t)1024)>>>(d_vertices,
                                                                                       d_faces,
                                                                                       d_group_id,
                                                                                       d_triangles,
                                                                                       triangles->size(), total_vertex_count,
                                                                                       d_index_lookup,
                                                                                       0,
                                                                                       d_vertex_index_out,
                                                                                       d_face_index_out);
    hipDeviceSynchronize();

    for (int i = 0; i < group_index; i++) {
        hipMemcpy(&vertex_index_out[i], &d_vertex_index_out[i], sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&face_index_out[i], &d_face_index_out[i], sizeof(int), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < group_index; i++) {
        hipMemcpy(vertex_out[i], &d_vertices[i * (triangles->size() + 3)], vertex_index_out[i] * sizeof(glm::vec3), hipMemcpyDeviceToHost);
        hipMemcpy(face_out[i], &d_faces[i * triangles->size()], face_index_out[i] * sizeof(glm::ivec3), hipMemcpyDeviceToHost);
    }

    hipFree(d_index_lookup);
    hipFree(d_vertex_index_out);
    hipFree(d_face_index_out);

    for (int i = 0; i < result.size(); i++) {
        result[i]->vertex.insert(result[i]->vertex.begin(), vertex_out[i], vertex_out[i] + vertex_index_out[i]);
        result[i]->index.insert(result[i]->index.begin(), face_out[i], face_out[i] + face_index_out[i]);
    }

    hipFree(d_vertices);
    hipFree(d_faces);

    for (int i = 0; i < group_index; i++) hipHostFree(&vertex_out[i]);
    for (int i = 0; i < group_index; i++) hipHostFree(&face_out[i]);
    hipHostFree(vertex_index_out);
    hipHostFree(face_index_out);

    return result;
}
