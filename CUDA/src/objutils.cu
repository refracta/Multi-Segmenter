#include "hip/hip_runtime.h"
#include "objutils.h"

__host__ __device__ void __calc_normal(face_t& face, vertex_t* vertices) {
    // 대응하는 정점 설정.
    vertex_t* v1 = &vertices[face.pi];
    vertex_t* v2 = &vertices[face.qi];
    vertex_t* v3 = &vertices[face.ri];

    // v2를 중심으로 한 방향 벡터 계산.
    vertex_t l1 = { v1->x - v2->x, v1->y - v2->y, v1->z - v2->z };
    vertex_t l2 = { v3->x - v2->x, v3->y - v2->y, v3->z - v2->z };

    // 외적을 통한 법선 벡터 계산.
    face.nx = l1.y * l2.z - l1.z * l2.y;
    face.ny = l1.z * l2.x - l1.x * l2.z;
    face.nz = l1.x * l2.y - l1.y * l2.x;

    // 정규화를 위한 법선 벡터의 크기 계산.
    float norm = sqrt(face.nx * face.nx + face.ny * face.ny + face.nz * face.nz);

    // 단위 법선 벡터 계산.
    face.nx /= norm;
    face.ny /= norm;
    face.nz /= norm;
}

__global__ void __calc_face_normals_cu(face_t* faces, vertex_t* vertices, size_t count) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    __calc_normal(faces[index], vertices);
}

void calc_face_normals(object_t& obj) {
    for (face_t& face : obj.faces) {
        __calc_normal(face, obj.vertices.data());
    }
}

void calc_face_normals_cu(object_t& obj) {
    object_dt d_obj = {
        thrust::device_vector<vertex_t>(obj.vertices),
        thrust::device_vector<face_t>(obj.faces)
    };

    size_t len_block = 1024;
    size_t len_grid = ceil((float)d_obj.faces.size() / len_block);

    __calc_face_normals_cu<<<len_grid, len_block>>>(thrust::raw_pointer_cast(d_obj.faces.data()),
                                                    thrust::raw_pointer_cast(d_obj.vertices.data()),
                                                    d_obj.faces.size());
    hipDeviceSynchronize();

    thrust::copy(d_obj.vertices.begin(), d_obj.vertices.end(), obj.vertices.begin());
    thrust::copy(d_obj.faces.begin(), d_obj.faces.end(), obj.faces.begin());
    hipDeviceSynchronize();
}
