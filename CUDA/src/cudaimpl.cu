#include "hip/hip_runtime.h"
#include "cudaheader.h"

__global__ void __foo_cuda(void** args) {
    int count = *(int*)args[0];
    float value = *(float*)args[1];

    #pragma omp parallel num_threads(4)
    {
        printf("OpenMP in CUDA! This should be printed 4 time.\n");
    }

    printf("Hello from CUDA Kernel (%d %d %d), (%d %d %d) with args: %d, %f!\n",
            blockIdx.x, blockIdx.y, blockIdx.z,
            threadIdx.x, threadIdx.y, threadIdx.z,
            count, value);
}
void (*foo_cuda)(void**) = __foo_cuda;

void kernel_call(void (*func)(void**), dim3 grid_dim, dim3 block_dim, void** args) {
    func<<<grid_dim, block_dim>>>(args);
}
